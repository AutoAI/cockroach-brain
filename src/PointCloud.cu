#include "hip/hip_runtime.h"
// PointCloud.cu

#include "PointCloud.hpp"

#ifdef __unix
#define fopen_s(pFile,filename,mode) ((*(pFile))=fopen((filename),(mode)))==NULL
#endif

PointCloud::PointCloud() {

}

PointCloud::PointCloud(size_t width, size_t height) {
	Width = width;
	Height = height;
	pc = new POINT3D[Width * Height];
}

PointCloud::~PointCloud() {
	
}

void PointCloud::fill(const unsigned char* image, const float* depth_map, const sl::zed::StereoParameters *param) {
	// allocate memory for a device copy of pc
	POINT3D* dev_pc;
	hipMalloc((void**)&dev_pc, Width * Height * sizeof(POINT3D));
	// copy host pc to device
	hipMemcpy(dev_pc, pc, Width * Height * sizeof(POINT3D), hipMemcpyHostToDevice);
	// fill
	parallelFill<<< 1, Width * Height >>>(image, depth_map, &Width, &(param->LeftCam.cx), &(param->LeftCam.cy), &(param->LeftCam.fx), &(param->LeftCam.fy));
	// copy device pc to host
	hipMemcpy(dev_pc, pc, Width * Height * sizeof(POINT3D), hipMemcpyDeviceToHost);
}

__global__ void parallelFill(const unsigned char* image, const float* depth_map, const int* Width, const float* cx, const float* cy, const float* fx, const float* fy) {
	int t = threadIdx.x;
	int j = t / Width;
	int i = t % Width;

	pc[t].setColor(&image[j * (Width * 4) + i * 4]);

	float depth = depth_map[t];
	depth /= 1000.f; // Convert to meters;

	pc[t].z = depth;
	pc[t].x = ((i - cx) * depth) / fx;
	pc[t].y = ((j - cy) * depth) / fy;
}

POINT3D PointCloud::Point(size_t i, size_t j) {
	return pc[i + Width * j];
}

POINT3D PointCloud::Point(size_t i) {
	return pc[i];
}

size_t PointCloud::GetNbPoints() {
	return Width * Height;
}

size_t PointCloud::GetWidth() {
	return Width;
}

size_t PointCloud::GetHeight() {
	return Height;
}
