#include "hip/hip_runtime.h"
// PointCloud.cu

#include "PointCloud.hpp"

#ifdef __unix
#define fopen_s(pFile,filename,mode) ((*(pFile))=fopen((filename),(mode)))==NULL
#endif

PointCloud::PointCloud() {

}

PointCloud::PointCloud(size_t width, size_t height) {
	Width = width;
	Height = height;
	pc.resize(Width * Height);
}

PointCloud::~PointCloud() {
	
}

void PointCloud::fill(const unsigned char* image, const float* depth_map, const sl::zed::StereoParameters *param) {
	parallelFill<<< 1, Width * Height >>>(image, depth_map, param->LeftCam.cx, param->LeftCam.cy, param->LeftCam.fx, param->LeftCam.fy);
}

__global__ void PointCloud::parallelFill(const unsigned char* image, const float* depth_map, const float* cx, const float* cy, const float* fx, const float* fy) {
	int t = threadIdx.x;
	int j = t / Width;
	int i = t % Width;

	pc[t].setColor(&image[j * (Width * 4) + i * 4]);

	float depth = depth_map[t];
	depth /= 1000.f; // Convert to meters;

	pc[t].z = depth;
	pc[t].x = ((i - cx) * depth) / fx;
	pc[t].y = ((j - cy) * depth) / fy;
}

POINT3D PointCloud::Point(size_t i, size_t j) {
	return pc[i + Width * j];
}

size_t PointCloud::GetNbPoints() {
	return pc.size();
}

size_t PointCloud::GetWidth() {
	return Width;
}

size_t PointCloud::GetHeight() {
	return Height;
}

void PointCloud::WritePCDFile(std::string path, bool verbose) {

	if (path.find(".pcd") == std::string::npos)
		path.append(".pcd");

	FILE *fich;
	fopen_s(&fich, path.c_str(), "w");

	fprintf(fich, "# .PCD v.7 - Point Cloud Data file format\n");
	fprintf(fich, "VERSION .7\n");
	fprintf(fich, "FIELDS x y z rgb\n");
	fprintf(fich, "SIZE 4 4 4 4\n");
	fprintf(fich, "TYPE F F F F\n");
	fprintf(fich, "COUNT 1 1 1 1\n");
	fprintf(fich, "WIDTH %d\n", Width);
	fprintf(fich, "HEIGHT %d\n", Height);
	fprintf(fich, "POINTS %d\n", Width * Height);
	fprintf(fich, "DATA ascii\n");

	if (verbose) printf("Saving PCD File ...");

	for (auto it = this->cbegin(); it != this->cend(); ++it)
		fprintf(fich, "%f %f %f %e\n", it->x, it->y, it->z, it->getColorFloat());

	if (verbose) printf("Done\n");

	fclose(fich);
}
