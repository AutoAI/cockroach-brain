#include "hip/hip_runtime.h"
// PointCloud.cu

#include "PointCloud.hpp"

#ifdef __unix
#define fopen_s(pFile,filename,mode) ((*(pFile))=fopen((filename),(mode)))==NULL
#endif

PointCloud::PointCloud() {

}

PointCloud::PointCloud(size_t width, size_t height) {
	Width = width;
	Height = height;
	pc = new POINT3D[Width * Height];
}

PointCloud::~PointCloud() {
	
}

__global__ void parallelFill(const unsigned char* image, const float* depth_map, const int* Width, POINT3D* dev_pc, const float* cx, const float* cy, const float* fx, const float* fy) {
	int t = threadIdx.x;
	int j = t / *Width;
	int i = t % *Width;

	dev_pc[t].r = image[j * (*Width * 4) + i * 4];
	dev_pc[t].g = image[j * (*Width * 4) + i * 4 + 1];
	dev_pc[t].b = image[j * (*Width * 4) + i * 4 + 2];

	float depth = depth_map[t];
	depth /= 1000.f; // Convert to meters;

	dev_pc[t].z = depth;
	dev_pc[t].x = ((i - *cx) * depth) / *fx;
	dev_pc[t].y = ((j - *cy) * depth) / *fy;
}

void PointCloud::fill(const unsigned char* image, const float* depth_map, const sl::zed::StereoParameters *param) {
	// allocate memory for a device copy of pc
	POINT3D* dev_pc;
	hipMalloc((void**)&dev_pc, Width * Height * sizeof(POINT3D));
	// copy host pc to device
	hipMemcpy(dev_pc, pc, Width * Height * sizeof(POINT3D), hipMemcpyHostToDevice);
	// fill
	parallelFill<<< 1, Width * Height >>>(image, depth_map, &Width, dev_pc, &(param->LeftCam.cx), &(param->LeftCam.cy), &(param->LeftCam.fx), &(param->LeftCam.fy));
	// copy device pc to host
	hipMemcpy(dev_pc, pc, Width * Height * sizeof(POINT3D), hipMemcpyDeviceToHost);
	// free allocated device memory
	hipFree((void*)dev_pc);

	for(int j = 0; j < Height; j++) {
		for(int i = 0; i < Height; i++) {
			std::cout << pc[i + Width * j]->r << '\t' << pc[i + Width * j]->g << '\t' << pc[i + Width * j]->b << '\t' << pc[i + Width * j]->x << '\t' << pc[i + Width * j]->y << '\t' << pc[i + Width * j]->z << '\t' << std::endl;
		}
	}
}

POINT3D PointCloud::Point(size_t i, size_t j) {
	return pc[i + Width * j];
}

POINT3D PointCloud::Point(size_t i) {
	return pc[i];
}

size_t PointCloud::GetNbPoints() {
	return Width * Height;
}

size_t PointCloud::GetWidth() {
	return Width;
}

size_t PointCloud::GetHeight() {
	return Height;
}
