///////////////////////////////////////////////////////////////////////////
//
// Copyright (c) 2015, STEREOLABS.
// 
// All rights reserved.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
// "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
// LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
// A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
// OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
// SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
// LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
// DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
// THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
///////////////////////////////////////////////////////////////////////////

//standard Include
#include <stdio.h>
#include <string.h>
#include <chrono>

//camera Include
#include <zed/Mat.hpp>
#include <zed/Camera.hpp>
#include <zed/utils/GlobalDefine.hpp>

//our point cloud generator and viewer.
#include "PointCloud.hpp"
#include "CloudViewer.hpp"

using namespace sl::zed;
using namespace std;

//main Loop

int main() {
	
    Camera* camera = new Camera(VGA,15.0);
    ERRCODE err = camera->init(MODE::PERFORMANCE, 0);

    int width = camera->getImageSize().width;
    int height = camera->getImageSize().height;

    // ERRCODE display
    cout << errcode2str(err) << endl;

    // Quit if an error occurred
    if (err != SUCCESS) {
        delete camera;
        return 1;
    }

    int key = ' ';

    Mat depth, imLeft;

    PointCloud *cloud = new PointCloud(width, height);
    CloudViewer *viewer = new CloudViewer();

    // application quits when user stikes 'q'
    while (key != 'q') {
        // DisparityMap filtering
		camera->setConfidenceThreshold(96); // parameter is reliability index ~[1,100] with 100 as no filtering

        // Get frames and launch the computation
        bool res = camera->grab(SENSING_MODE::RAW);

        depth = camera->retrieveMeasure(MEASURE::DEPTH);
        imLeft = camera->retrieveImage(SIDE::LEFT);
        cloud->fill(imLeft.data, (float*) depth.data, camera->getParameters());
        viewer->AddData(cloud);

        // Update the value of key so that we can quit when the user strikes 'q'
        key = viewer->getKey();
    }

    delete camera;
    delete cloud;
    delete viewer;
    return 0;
}
